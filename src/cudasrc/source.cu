#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <source.cuh>
#include <>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <c:/Users/mtf_d/Desktop/src/IUnclassifiedPoints.h>
#include "c:/Users/mtf_d/Desktop/src/IPoint.h"
#include "c:/Users/mtf_d/Desktop/src/IUnclassifiedPoints.h"
#include "c:/Users/mtf_d/Desktop/src/StackedPoints.h"
#include <c:/Users/mtf_d/Desktop/src/UnclassifiedPoints.h>
//#include "c:/Users/mtf_d/Desktop/src/PointVector.h"


namespace mcc{

	#include <cstdio>
		inline void GPUassert(hipError_t code, char * file, int line, bool Abort = true)
		{
			if (code != 0) {
				fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
				if (Abort) exit(code);
			}
		}

	#define GPUerrchk(ans) { GPUassert((ans), __FILE__, __LINE__); }


	
	__global__ void testkernel(double *A, int arraySize) {
		int threadID = blockDim.x * blockIdx.x + threadIdx.x;
		printf("threadid : %d", threadID);
		if (threadID < 10) {
			printf("threadid : %d", threadID);
		}

		/*if (threadID < 10) {
			for (int i = 0, int c = 0; i < 2; i++) {
				for (int j = 0; j < 25;) {
					if (c % 5 == 0) {
						printf("X : %f	i: %d	", A[c], c);
					}
					if (c % 5 == 1) {
						printf("Y : %f	i: %d	", A[c], c);
					}
					if (c % 5 == 2) {
						printf("Z : %f	i: %d	", A[c], c);
					}
					if (c % 5 == 3) {
						printf("S : %f	i: %d	", A[c], c);
					}
					if (c % 5 == 4) {
						printf("V--->> : %f	i: %d\n", A[c], c);
						j++;
					}
					c++;
				}
			}
		}*/



		/*if (threadID < 10) {
			/*for (int d = 0; d < 5; d++) {
				A[0] += A[0];
				printf("in device value of -->>> %f\n", A[0]);
			}*/
			/*printf("%f", A[0][0][0]);
			int size = (int)A[0][0][3];
			for (int j = 0; j < size; j++)
			{
				printf("%f ", A[threadID][j][0]);
				printf("%f ", A[threadID][j][1]);
				printf("%f ", A[threadID][j][2]);
				printf("%f \n", A[threadID][j][3]);
			}

		}*/
			
	}
	__global__ void vectorAdditionKernel(double* A, double* B, double* C, int arraySize) {
		// Get thread ID.
		int threadID = blockDim.x * blockIdx.x + threadIdx.x;

		// Check if thread is within array bounds.
		if (threadID < arraySize) {
			// Add a and b.
			C[threadID] = A[threadID] + B[threadID];
			printf("global ici");
		}
	}
	__global__ void vkadd(double* A, double* B, int arraySize) {
		// Get thread ID.
		int threadID = blockDim.x * blockIdx.x + threadIdx.x;

		// Check if thread is within array bounds.
		if (threadID < arraySize) {
			// Add a and b.
			printf("global ici vkADD");
		}
	}

	void source::kernel(double* A, double* B, double* C, int arraySize) {

		/*int numElements = 5000000;
		size_t size = numElements * sizeof(float);
		float *h_A = (float *)malloc(size);

		hipDeviceptr_t d_X;
		hipMalloc((void**)&d_X, size * sizeof(double));*/

		// Initialize device pointers.

		double *xa = (double*)malloc(sizeof(double) * 3);
		xa[0] = 0;
		xa[1] = 1;
		xa[2] = 2;
		double *xb = (double*)malloc(sizeof(double) * 3);
		xb[0] = 0;
		xb[1] = 1;
		xb[2] = 2;
		double*xx;
		double*xy;
		hipMalloc(&xx, 3 * sizeof(double));
		hipMalloc(&xy, 3 * sizeof(double));

		hipMemcpy(xx, xa, 3 * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(xy, xb, 3 * sizeof(double), hipMemcpyHostToDevice);

		double* d_A, *d_B, *d_C;

		// Allocate device memory.
		hipMalloc(&d_A, arraySize * sizeof(double));
		hipMalloc(&d_B, arraySize * sizeof(double));
		hipMalloc(&d_C, arraySize * sizeof(double));

		// Transfer arrays a and b to device.
		hipMemcpy(d_A, A, 5 * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(d_B, B, arraySize * sizeof(double), hipMemcpyHostToDevice);
		//printf("kernel deger icinde d_A degeri : %f", d_A);

		// Calculate blocksize and gridsize.
		dim3 blockSize(512, 1, 1);
		dim3 gridSize(512 / arraySize + 1, 1);

		// Launch CUDA kernel.

		vectorAdditionKernel << <1,1 >> > (xx, xy, d_C, 3);
		//vectorAdditionKernel << <gridSize, blockSize >> > (d_A, d_B, d_C, arraySize);

		// Copy result array c back to host memory.
		hipMemcpy(C, d_C, arraySize * sizeof(double), hipMemcpyDeviceToHost);

		/*int x = 0;
		scanf("%d",&x);
		printf("cuda kernel matris toplam�\n");*/

	}

	void source::clas(IUnclassifiedPoints & points) {
		std::cout << "cu icine points atma, say�s�:--> " << points.count() << std::endl;
	}

	void source::test(double ***A, int Asize) {
		int arraySize = 3;
		double *xa = (double*)malloc(sizeof(double) * 3);
		xa[0] = 0;
		xa[1] = 1;
		xa[2] = 2;
		double *xb = (double*)malloc(sizeof(double) * 3);
		xb[0] = 0;
		xb[1] = 1;
		xb[2] = 2;
		double*xx;
		double*xy;
		hipMalloc(&xx, 3 * sizeof(double));
		hipMalloc(&xy, 3 * sizeof(double));

		hipMemcpy(xx, xa, 3 * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(xy, xb, 3 * sizeof(double), hipMemcpyHostToDevice);

		double* d_A, *d_B, *d_C;

		// Allocate device memory.
		hipMalloc(&d_A, arraySize * sizeof(double));
		hipMalloc(&d_B, arraySize * sizeof(double));
		hipMalloc(&d_C, arraySize * sizeof(double));

		// Transfer arrays a and b to device.
		//hipMemcpy(d_A, A, 5 * sizeof(double), hipMemcpyHostToDevice);
		//hipMemcpy(d_B, B, arraySize * sizeof(double), hipMemcpyHostToDevice);
		//printf("kernel deger icinde d_A degeri : %f", d_A);

		// Calculate blocksize and gridsize.
		dim3 blockSize(512, 1, 1);
		dim3 gridSize(512 / arraySize + 1, 1);

		// Launch CUDA kernel.

		testkernel << <1, 1 >> > (xx, 3);
		/*int val = 0;
		for (int i = 0; i < Asize; i++) {
			val += A[i][0][3];
		}
		printf("val: %d\n", val);
		int d = val * 5;
		printf("val*5: %d\n", d);
		double *X = (double*)malloc(sizeof(double)*d);

		double a = 0, b = 0;

		for (int i = 0,int c = 0; i < Asize; i++) {
			int tSize = A[i][0][3];
			for (int j = 0; j < tSize;) {
				if (c % 5 == 0) {
					X[c] = A[i][j][0];
					//printf("X : %f	i: %d	", X[c], c);
				}
				if (c % 5 == 1) {
					X[c] = A[i][j][1];
					//printf("Y : %f	i: %d	", X[c], c);
				}
				if (c % 5 == 2) {
					X[c] = A[i][j][2];
					//printf("Z : %f	i: %d	", X[c], c);
				}
				if (c % 5 == 3) {
					X[c] = A[i][j][3];
					//printf("S : %f	i: %d	", X[c], c);
				}
				if (c % 5 == 4) {
					a++;
					X[c] = b;
					//printf("V--->> : %f	i: %d\n", X[c], c);
					if (a == A[i][j][3]) {
						//printf("V--->> : %f	i: %d\n", X[c], c);
						a = 0;
						b++;
					}
					j++;
				}
				//printf("	a: %f", a);
				//printf("	A: %f\n", A[i][j][3]);
				if (c == d)
					break;
				c++;
			}
		}
		printf("%f\n", X[d-1]);

		double *xa = (double*)malloc(sizeof(double) * 3);
		xa[0] = 0;
		xa[1] = 1;
		xa[2] = 2;
		double *xb = (double*)malloc(sizeof(double) * 3);
		xb[0] = 0;
		xb[1] = 1;
		xb[2] = 2;
		double*xx;
		double*xy;
		hipMalloc(&xx, 3 * sizeof(double));
		hipMalloc(&xy, 3 * sizeof(double));
		double *d_C;
		hipMalloc(&d_C, 3 * sizeof(double));

		hipMemcpy(xx, xa, 3 * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(xy, xb, 3 * sizeof(double), hipMemcpyHostToDevice);

		vkadd << <10, 10 >> > (xx, xy, d_C, 3);

		/*double *d_A;

		hipMalloc(&d_A, 3 * sizeof(double));

		hipMemcpy(d_A, xa, 3 * sizeof(double), hipMemcpyHostToDevice);
		
		testkernel <<<10, 10 >>> (d_A, 3);
		
		for (int i = 0, int c = d-120; i < 2; i++) {
			int tSize = A[i][0][3];
			for (int j = 0; j < tSize;) {
				if (c % 5 == 0) {
					printf("X : %f	i: %d	", X[c], c);
				}
				if (c % 5 == 1) {
					printf("Y : %f	i: %d	", X[c], c);
				}
				if (c % 5 == 2) {
					printf("Z : %f	i: %d	", X[c], c);
				}
				if (c % 5 == 3) {
					printf("S : %f	i: %d	", X[c], c);
				}
				if (c % 5 == 4) {
					printf("V--->> : %f	i: %d\n", X[c], c);
					j++;
				}
				c++;
			}
		}

		for (int i = d-25; i < d; i++)
		{
			int tSize = A[i][0][3];
			for (int j = 0; j < tSize; j++)
			{
				//std::cout << A[i][j][0] << " ";
				printf("%f ", A[i][j][0]);
				//std::cout << A[i][j][1] << " ";
				printf("%f ", A[i][j][1]);
				//std::cout << A[i][j][3] << " ";
				printf("%f ", A[i][j][2]);
				//std::cout << A[i][j][4] << " ";
				printf("%f ", A[i][j][3]);
				std::cout << std::endl;
			}
		}*/
























		/*for (int i = 0; i < 2; i++)
		{
			int tSize = A[i][0][3];
			for (int j = 0; j < tSize; j++)
			{
				//std::cout << A[i][j][0] << " ";
				printf("%f ", A[i][j][0]);
				//std::cout << A[i][j][1] << " ";
				printf("%f ", A[i][j][1]);
				//std::cout << A[i][j][3] << " ";
				printf("%f ", A[i][j][2]);
				//std::cout << A[i][j][4] << " ";
				printf("%f ", A[i][j][3]);
				std::cout << std::endl;
			}
		}*/
		/*for (int i = 0, int c = 0; i < 2; i++) {
			int tSize = A[i][0][3];
			for (int j = 0; j < tSize;) {
				if (c % 5 == 0) {
					printf("X : %f	i: %d	", X[c], c);
				}
				if (c % 5 == 1) {
					printf("Y : %f	i: %d	", X[c], c);
				}
				if (c % 5 == 2) {
					printf("Z : %f	i: %d	", X[c], c);
				}
				if (c % 5 == 3) {
					printf("S : %f	i: %d	", X[c], c);
				}
				if (c % 5 == 4) {
					printf("V--->> : %f	i: %d\n", X[c], c);
					j++;
				}
				c++;
			}
		}*/





		//printf("X[0]: %f --- A: %f", X[0]);

		/*for (int i = 0; i < 10; i++) {
			if (i % 5 == 0) {
				printf("bu x: %d", X[i]);
				printf("	bu da Ax: %d\n", A[i][0][0]);
			}
			if (i % 5 == 1) {
				printf("bu y:%d", X[i]);
				printf("	bu da Ay: %d\n", A[i][0][1]);
			}
			if (i % 5 == 2) {
				printf("bu z:%d", X[i]);
				printf("	bu da Az: %d\n", A[i][0][2]);
			}
			if (i % 5 == 3) {
				printf("bu size:%d", X[i]);
				printf("	bu da Asize: %d\n", A[i][0][3]);
			}
			if (i % 5 == 4) {
				printf("bu val: %d\n", X[i]);
				//printf("bu da Ax: ", A[i][0][4]);
			}
		}*/

		/*for (int i = 0; i < 10; i++)
		{
			for (int j = 0; j < 10; j++)
			{
				//std::cout << A[i][j][0] << " ";
				printf("%f ", A[i][j][0]);
				//std::cout << A[i][j][1] << " ";
				printf("%f ", A[i][j][1]);
				//std::cout << A[i][j][3] << " ";
				printf("%f ", A[i][j][2]);
				//std::cout << A[i][j][4] << " ";
				printf("%f ", A[i][j][3]);
				std::cout << std::endl;
			}
		}*/




		/*double ***d_A;
		hipMalloc(&d_A, Asize * sizeof(double));

		for (int i = 0; i < Asize; i++) {
			int tSize = A[i][0][3];
			hipMalloc(&d_A[i], tSize * sizeof(double));
			for (int j = 0; j < tSize; j++) {
				hipMalloc(&d_A[i][j], 4 * sizeof(double));
				hipMemcpy(&d_A[i][j], A[i][j], 4 * sizeof(double), hipMemcpyHostToDevice);
			}
		}
		hipMemcpy(&d_A, A, Asize, hipMemcpyHostToDevice);
		testkernel << <10, 10 >> > (d_A, 3);*/


		/*printf("malloc begin");
		for (int i = 0; i < Asize; i++) {
			int twoSize = A[i][0][3];
			for (int j = 0; j < twoSize; j++) {
				GPUerrchk(hipMalloc(&A[i][j], 4 * sizeof(double)));
			}
		}
		printf("malloc comp");

		for (int i = 0; i < Asize; i++) {
			int s = A[i][0][3];
			GPUerrchk(hipMalloc(&A[i], s * sizeof(double)));
			GPUerrchk(hipMemcpy(A[i], A[i], s * sizeof(double), hipMemcpyHostToDevice));
		}
		printf("memcpy comp ");

		

		GPUerrchk(hipMalloc(&d_A, Asize * sizeof(double)));
		GPUerrchk(hipMemcpy(d_A, A, Asize * sizeof(double), hipMemcpyHostToDevice));
		testkernel << <10, 10 >> > (d_A, 3);*/
		//double ***d_A;
		/*for (int i = 0; i < Asize; i++) {
			int twoSize = A[i][0][3];
			//printf("ilk d�ng� size: %d",twoSize);
			for (int j = 0; j < twoSize; j++) {
				//int arrSize = A[i][0][0];
				//printf("ikinci dongu");
				hipMalloc(&d_A[i][j], 4 * sizeof(double));
				hipMemcpy(d_A[i][j], A[i][j], 4 * sizeof(double),hipMemcpyHostToDevice);
			}
		}
		for (int j = 0; j < Asize; j++) {
			int twoSize = A[j][0][3];
			for (int i = 0; i < twoSize; i++) {
				hipMalloc(&d_A[i], twoSize * sizeof(double));
			}
		}

		hipMalloc(&d_A, Asize * sizeof(double));*/

		/*
		for (int a = 0; a < Asize; a++) {
			int twoArrSize = A[a][0][3];
			//printf("ilk d�ng� size: %d", twoArrSize);
			for (int b = 0; b < twoArrSize; b++) {
				//printf("ikinci dongu");
				hipMemcpy(d_A[a][b], A[a][b], 4 * sizeof(double), hipMemcpyHostToDevice);
				//printf("atama sonras� d_A: %f", d_A[0][0][3]);
			}
		}*/
		


		/*int val = 0;
		for (int i = 0; i < Asize; i++) {
			val += A[i][0][3];
			//val += val;
		}
		printf("val %d\n", val);
		int a[10];
		for (int i = 1; i < 11; i++) {
			a[i] = i;
		}
		int deg = 0;
		for (int i = 1; i < 11; i++) {
			deg += a[i];
			printf("%d\n", a[i]);
		}
		printf("deg: %d\n", deg);*/



		//printf("cu icinde test %f", A[0][0][0]);
		//std::cout << "cu icinde test" << A[0] << std::endl;
		/*
		double* d_A;
		double* P;
		P = A;
		hipMalloc((void**)&d_A,sizeof(double));
		hipMemcpy(d_A, P, 1 * sizeof(double), hipMemcpyHostToDevice);

		test << <2,2 >> > (d_A);
		*/
		
		/*for (int i = 0; i < 10; i++)
		{
			for (int j = 0; j < 10; j++)
			{
				//std::cout << A[i][j][0] << " ";
			printf("%f ", A[i][j][0]);
			//std::cout << A[i][j][1] << " ";
			printf("%f ", A[i][j][1]);
			//std::cout << A[i][j][3] << " ";
			printf("%f ", A[i][j][2]);
			//std::cout << A[i][j][4] << " ";
			printf("%f ", A[i][j][3]);
			}
			std::cout << std::endl;
		}*/
		/*for (int i = 0; i < 10; i++)
		{
			for (int j = 0; j < 10; j++)
			{
				//std::cout << A[i][j][0] << " ";
				printf("%f ", A[i][j][0]);
				//std::cout << A[i][j][1] << " ";
				printf("%f ", A[i][j][1]);
				//std::cout << A[i][j][3] << " ";
				printf("%f ", A[i][j][2]);
				//std::cout << A[i][j][4] << " ";
				printf("%f ", A[i][j][3]);
				std::cout << std::endl;
			}
		}*/

		/*printf("A dizisi boyutu : %d\n", Asize);


		// Initialize device pointers.
		//double*** d_A[Asize];
		double*** d_A = new double**[Asize];
		for (int i = 0; i < Asize; i++) {
			int twoSize = A[i][0][3];
			d_A[i] = new double*[twoSize];
			for (int j = 0; j < twoSize; j++) {
				d_A[i][j] = new double[4];
			}
		}*/

		//double ***d_A = A;
		/*printf("d_A deger : %f\n", d_A[0][0][3]);
		printf("A value : %f", A[0][0][3]);
		d_A[0][0][3] = 5;
		printf("d_A deger : %f\n", d_A[0][0][3]);
		printf("A value : %f\n", A[0][0][3]);
		*/
		// Allocate device memory.
		//hipMalloc((void****)&d_A, arraySize * sizeof(double));

		/*for (int j = 0; j < Asize; j++) {
			for (int i = A[0][0][0]; i < Asize; i++) {
				hipMalloc(&A[i][j], Asize * sizeof(float));
			}
		}*/
		/*printf("allocate started");
		for (int i = 0; i < Asize; i++) {
			int twoSize = A[i][0][3];
			//printf("ilk d�ng� size: %d",twoSize);
			for (int j = 0; j < twoSize; j++) {
				//int arrSize = A[i][0][0];
				//printf("ikinci dongu");
				hipMalloc(&d_A[i][j], 4 * sizeof(double));
			}
		}*/
		//printf("allocate completed");
		/*
		int x = 0;
		scanf("%d", &x);
		*/
		/*for (int i = 0; i < n; i++) {
			hipMemcpy(temph[i], a[i], n * sizeof(float), hipMemcpyHostToDevice);
		}*/

		//printf("%d", Asize);

		/*for (int a = 0; a < Asize; a++) {
			int twoArrSize = A[a][0][3];
			//printf("ilk d�ng� size: %d", twoArrSize);
			for (int b = 0; b < twoArrSize; b++) {
				//printf("ikinci dongu");
				hipMemcpy(d_A[a][b], A[a][b], 4 * sizeof(double), hipMemcpyHostToDevice);
				//printf("atama sonras� d_A: %f", d_A[0][0][3]);
			}
		}
		printf("copy completed");*/


		// Transfer arrays a and b to device.
		//hipMemcpy(d_A, A, arraySize * sizeof(double), hipMemcpyHostToDevice);

		// Calculate blocksize and gridsize.
		/*dim3 blockSize(512, 1, 1);
		dim3 gridSize(512 / 3 + 1, 1);

		// Launch CUDA kernel.
		testkernel << <blockSize,gridSize >> > (d_A,3);*/
		printf("kernel alti");
}
}