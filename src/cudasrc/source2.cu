#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <source2.cuh>
#include <>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <c:/Users/mtf_d/Desktop/src/IUnclassifiedPoints.h>
#include "c:/Users/mtf_d/Desktop/src/IPoint.h"
#include "c:/Users/mtf_d/Desktop/src/IUnclassifiedPoints.h"
#include "c:/Users/mtf_d/Desktop/src/StackedPoints.h"
#include <c:/Users/mtf_d/Desktop/src/UnclassifiedPoints.h>
//#include "c:/Users/mtf_d/Desktop/src/PointVector.h"


namespace mcc {
	__global__ void basari(double ***A, int Asize) {
		int threadID = blockDim.x * blockIdx.x + threadIdx.x;
		//printf("thread id : %d", threadID);
		//printf("	%f\n", A[threadID][threadID][threadID]);
		for (int i = threadID; i < Asize; i++)
		{
			int tSize = A[i][0][3];
			for (int j = 0; j < tSize; j++)
			{
				//std::cout << A[i][j][0] << " ";
				printf("X:%f %d", A[i][j][0],threadID);
				//std::cout << A[i][j][1] << " ";
				printf("Y:%f %d", A[i][j][1], threadID);
				//std::cout << A[i][j][3] << " ";
				printf("Z:%f %d", A[i][j][2], threadID);
				//std::cout << A[i][j][4] << " ";
				printf("S:%f %d", A[i][j][3], threadID);
				printf("\n");
			}
			printf("\n");
		}
	}

	__global__ void luDeSpline(double***A, double***mtx_v, double***mtx_l, double***cells) {

	}
	__global__ void cels(double ***cells) {
		int threadID = blockDim.x * blockIdx.x + threadIdx.x;
		printf("tid : %d cell.x : %f cell.y : %f\n", threadID, cells[threadID][0][1], cells[threadID][0][2]);
	}

	int source2::bos(double ***A, double ***cellsize, int Asize) {
		//allocate control_points
		std::cout << "begin d_C" << std::endl;
		double*** h_c = (double***)malloc(Asize * sizeof(double**));
		for (int i = 0; i < Asize; i++) {
			int twoSize = A[i][0][3];
			h_c[i] = (double**)malloc(twoSize * sizeof(double*));
			for (int j = 0; j < twoSize; j++) {
				hipMalloc((void**)&h_c[i][j], 4 * sizeof(double));
				hipMemcpy(h_c[i][j], A[i][j], 4 * sizeof(double), hipMemcpyHostToDevice);
			}
		}
		double ***h_c1 = (double ***)malloc(Asize * sizeof(double **));
		for (int i = 0; i < Asize; i++) {
			int twoSize = A[i][0][3];
			hipMalloc((void***)&(h_c1[i]), twoSize * sizeof(double*));
			hipMemcpy(h_c1[i], h_c[i], twoSize * sizeof(double*), hipMemcpyHostToDevice);
		}
		
		double*** d_c;
		hipMalloc((void****)&d_c, Asize * sizeof(double**));
		hipMemcpy(d_c, h_c1, Asize * sizeof(double**), hipMemcpyHostToDevice);
		std::cout << "end d_C" << std::endl;
		

		//allocate mtx_v
		std::cout << "begin mtx_v" << std::endl;
		double*** mtx_v1 = (double***)malloc(Asize * sizeof(double**));
		for (int i = 0; i < Asize; i++) {
			int twoSize = A[i][0][3]+3;
			mtx_v1[i] = (double**)malloc(twoSize * sizeof(double*));
			for (int j = 0; j < twoSize; j++) {
				hipMalloc((void**)&mtx_v1[i][j], 1 * sizeof(double));
			}
		}
		double ***mtx_v = (double ***)malloc(Asize * sizeof(double **));
		for (int i = 0; i < Asize; i++) {
			int twoSize = A[i][0][3];
			hipMalloc((void***)&(mtx_v[i]), twoSize * sizeof(double*));
			hipMemcpy(mtx_v[i], h_c[i], twoSize * sizeof(double*), hipMemcpyHostToDevice);
		}
		double*** d_mtx_v;
		hipMalloc((void****)&d_mtx_v, Asize * sizeof(double**));
		hipMemcpy(d_mtx_v, mtx_v, Asize * sizeof(double**), hipMemcpyHostToDevice);
		std::cout << "end mtx_v" << std::endl;


		//allocate mtx_l
		std::cout << "begin mtx_l" << std::endl;
		double*** mtx_l1 = (double***)malloc(Asize * sizeof(double**));
		for (int i = 0; i < Asize; i++) {
			int twoSize = A[i][0][3] + 3;
			mtx_l1[i] = (double**)malloc(twoSize * sizeof(double*));
			for (int j = 0; j < twoSize; j++) {
				hipMalloc((void**)&mtx_l1[i][j], twoSize * sizeof(double));
			}
		}
		double ***mtx_l = (double ***)malloc(Asize * sizeof(double **));
		for (int i = 0; i < Asize; i++) {
			int twoSize = A[i][0][3] + 3;
			hipMalloc((void***)&(mtx_v[i]), twoSize * sizeof(double*));
			hipMemcpy(mtx_l1[i], mtx_l[i], twoSize * sizeof(double*), hipMemcpyHostToDevice);
		}
		double*** d_mtx_l;
		hipMalloc((void****)&d_mtx_l, Asize * sizeof(double**));
		hipMemcpy(d_mtx_l, mtx_l, Asize * sizeof(double**), hipMemcpyHostToDevice);
		std::cout << "end mtx_l" << std::endl;

		//allocate cells
		std::cout << "begin cells" << std::endl;
		double*** cells1 = (double***)malloc(Asize * sizeof(double**));
		for (int i = 0; i < Asize; i++) {
			int twoSize = cellsize[i][0][0];
			cells1[i] = (double**)malloc(twoSize * sizeof(double*));
			for (int j = 0; j < twoSize; j++) {
				hipMalloc((void**)&cells1[i][j], 3 * sizeof(double));
				hipMemcpy(cells1[i][j], cellsize[i][j], 3 * sizeof(double), hipMemcpyHostToDevice);
			}
		}
		double ***cells = (double ***)malloc(Asize * sizeof(double **));
		for (int i = 0; i < Asize; i++) {
			int twoSize = cellsize[i][0][0];
			hipMalloc((void***)&(mtx_v[i]), twoSize * sizeof(double*));
			hipMemcpy(cells[i], cells1[i], twoSize * sizeof(double*), hipMemcpyHostToDevice);
		}
		double*** d_cells;
		hipMalloc((void****)&d_cells, Asize * sizeof(double**));
		hipMemcpy(d_cells, cells, Asize * sizeof(double**), hipMemcpyHostToDevice);
		std::cout << "end cells" << std::endl;
























		/*double ***d_A;
		A[0][0][0] = 999;
		printf("dA 0 0 0 : %f", A[0][0][0]);
		hipMalloc(&d_A, Asize * sizeof(double));
		int x;
		scanf("%d", &x);
		printf("allocate started");
		for (int i = 0; i < Asize; i++) {
			int tSize = A[i][0][3];
			printf("2");
			hipMalloc(&d_A[i], tSize * sizeof(double));
			printf("3");
			for (int j = 0; j < tSize; j++) {
				printf("4");
				hipMalloc(&d_A[i][j], 4 * sizeof(double));
				hipMemcpy(d_A[i][j], A[i][j], 4 * sizeof(double), hipMemcpyHostToDevice);
			}
		}
		printf("allocate completed");
		hipMemcpy(&d_A, A, Asize, hipMemcpyHostToDevice);
		kk << <10, 10 >> > (d_A, 3);*/

		/*double **B;
		hipMalloc(&B, 2 * sizeof(double));
		hipMalloc(&B[0], 2 * sizeof(double));
		hipMalloc(&B[1], 2 * sizeof(double));
		double *C = new double[2];
		C[0] = 1;
		C[1] = 2;
		double *D = new double[2];
		D[0] = 2;
		D[1] = 3;
		hipMemcpy(B[0], C, 2 * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(B[1], D, 2 * sizeof(double), hipMemcpyHostToDevice);
		aa << <2, 2 >> > (B);*/
		/*double *C = new double[2];
		C[0] = 1;
		C[1] = 2;
		double *D = new double[2];
		D[0] = 3;
		D[1] = 4;
		double **B;
		hipMalloc(&B[0], 2 * sizeof(double));
		hipMalloc(&B[1], 2 * sizeof(double));
		hipMemcpy(B[0], C, 2 * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(B[1], D, 2 * sizeof(double), hipMemcpyHostToDevice);
		hipMalloc(&B, 2 * sizeof(double));
		hipMemcpy(B, B, 2 * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(B, B, 2 * sizeof(double), hipMemcpyHostToDevice);
		aa << <2, 2 >> > ();*/







		/*int val = 0;
		for (int i = 0; i < Asize; i++) {
			val += A[i][0][3];
		}
		printf("val: %d\n", val);
		int d = val * 5;
		printf("val*5: %d\n", d);
		double *X = (double*)malloc(sizeof(double)*d);

		double a = 0, b = 0;

		for (int i = 0, int c = 0; i < Asize; i++) {
			int tSize = A[i][0][3];
			for (int j = 0; j < tSize;) {
				if (c % 5 == 0) {
					X[c] = A[i][j][0];
					//printf("X : %f	i: %d	", X[c], c);
				}
				if (c % 5 == 1) {
					X[c] = A[i][j][1];
					//printf("Y : %f	i: %d	", X[c], c);
				}
				if (c % 5 == 2) {
					X[c] = A[i][j][2];
					//printf("Z : %f	i: %d	", X[c], c);
				}
				if (c % 5 == 3) {
					X[c] = A[i][j][3];
					//printf("S : %f	i: %d	", X[c], c);
				}
				if (c % 5 == 4) {
					a++;
					X[c] = b;
					//printf("V--->> : %f	i: %d\n", X[c], c);
					if (a == A[i][j][3]) {
						//printf("V--->> : %f	i: %d\n", X[c], c);
						a = 0;
						b++;
					}
					j++;
				}
				//printf("	a: %f", a);
				//printf("	A: %f\n", A[i][j][3]);
				if (c == d)
					break;
				c++;
			}
		}
		printf("%f\n", X[d - 1]);
		double *d_A;

		hipMalloc(&d_A, d * sizeof(double));

		hipMemcpy(d_A, X, d * sizeof(double), hipMemcpyHostToDevice);

		kk << <2,3 >> > (d_A, 3);

		/*double***d_AA;

		for (int j = 0; j < Asize; j++) {
			for (int i = A[0][0][0]; i < Asize; i++) {
				hipMalloc(&A[i][j], Asize * sizeof(float));
			}
		}
		printf("allocate started");*/
		/*for (int i = 0; i < Asize; i++) {
			int twoSize = A[i][0][3];
			//printf("ilk d�ng� size: %d",twoSize);
			for (int j = 0; j < twoSize; j++) {
				//int arrSize = A[i][0][0];
				//printf("ikinci dongu");
				hipMalloc(&d_AA[i][j], 4 * sizeof(double));
			}
		}*/
		
		/*double*** d_AAA = new double**[Asize];
		for (int i = 0; i < Asize; i++) {
			int twoSize = A[i][0][3];
			d_AAA[i] = new double*[twoSize];
			for (int j = 0; j < twoSize; j++) {
				d_AAA[i][j] = new double[4];
			}
		}
		for (int i = 0; i < Asize; i++) {
			int twoSize = A[i][0][3];
			for (int j = 0; j < twoSize; j++) {
				hipMalloc(&d_AAA[i][j], 4 * sizeof(double));
			}
		}
		printf("allocate completed");

		printf("%d", Asize);

		for (int a = 0; a < Asize; a++) {
			int twoArrSize = A[a][0][3];
			//printf("ilk d�ng� size: %d", twoArrSize);
			for (int b = 0; b < twoArrSize; b++) {
				//printf("ikinci dongu");
				hipMemcpy(d_AAA[a][b], A[a][b], 4 * sizeof(double), hipMemcpyHostToDevice);
				//printf("atama sonras� d_A: %f", d_A[0][0][3]);
			}
		}
		printf("copy completed");

		kk << <2, 2 >> > (d_AAA, 3);*/

		return 1;
	}
}

