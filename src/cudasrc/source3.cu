#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <source3.cuh>
#include <>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <c:/Users/mtf_d/Desktop/src/IUnclassifiedPoints.h>
#include "c:/Users/mtf_d/Desktop/src/IPoint.h"
#include "c:/Users/mtf_d/Desktop/src/IUnclassifiedPoints.h"
#include "c:/Users/mtf_d/Desktop/src/StackedPoints.h"
#include <c:/Users/mtf_d/Desktop/src/UnclassifiedPoints.h>
#include <windows.h>
#include <cstdio>
#include <ctime>
#include <hiprand.h>

//#include "c:/Users/mtf_d/Desktop/src/PointVector.h"

namespace mcc
{
	inline void GPUassert(hipError_t code, char *file, int line, bool Abort = true)
	{
		if (code != 0)
		{
			fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
			if (Abort)
				exit(code);
		}
	}

#define GPUerrchk(ans)                        \
	{                                         \
		GPUassert((ans), __FILE__, __LINE__); \
	}

	__global__ void basarii(double ***A, int Asize)
	{
		int threadID = blockDim.x * blockIdx.x + threadIdx.x;
		//printf("thread id : %d", threadID);
		//printf("	%f\n", A[threadID][threadID][threadID]);
		for (int i = threadID; i < Asize; i++)
		{
			int tSize = A[i][0][3];
			for (int j = 0; j < tSize; j++)
			{
				//std::cout << A[i][j][0] << " ";
				printf("X:%f %d", A[i][j][0], threadID);
				//std::cout << A[i][j][1] << " ";
				printf("Y:%f %d", A[i][j][1], threadID);
				//std::cout << A[i][j][3] << " ";
				printf("Z:%f %d", A[i][j][2], threadID);
				//std::cout << A[i][j][4] << " ";
				printf("S:%f %d", A[i][j][3], threadID);
				printf("\n");
			}
			printf("\n");
		}
	}

	/*__global__ void cel(double ***A)
	{
		int threadID = blockDim.x * blockIdx.x + threadIdx.x;
		if (threadID == 0)
		{
			printf("in global");
		}
		int t = A[threadID][0][3];
		/*for (int j = 0; j < t; j++) {
			//std::cout << A[i][j][0] << " ";
			printf("X:%f %d", A[threadID][j][0], threadID);
			//std::cout << A[i][j][1] << " ";
			printf("Y:%f %d", A[threadID][j][1], threadID);
			//std::cout << A[i][j][3] << " ";
			printf("Z:%f %d", A[threadID][j][2], threadID);
			//std::cout << A[i][j][4] << " ";
			printf("S:%f %d", A[threadID][j][3], threadID);
			printf("\n");
			}*/
	//printf("global ici double pointer dec\n");
	/*double **res = new double *[t];
		for (int i = 0; i < t; i++)
		{
			res[i] = new double[4];
		}
		//printf("global ici double pointer id\n");
		for (int i = 0; i < t; i++)
		{
			for (int j = 0; j < 4; j++)
			{
				res[i][j] = A[threadID][i][j];
			}
		}
		for (int i = 0; i < t; i++)
		{
			printf("global ici res x : %f", res[i][0]);
			printf("global ici res y : %f", res[i][1]);
			printf("global ici res z : %f", res[i][2]);
			printf("global ici res s : %f", res[i][3]);
			printf("threadID : %d", threadID);
			printf("\n");
		}

		printf("global	");
		printf("%d\n", threadID);
		printf("cells x: %f cells y: %f", cells[threadID][0][1], cells[threadID][0][2]);*/
	/*int twos = cells[threadID][0][0];
		printf("global ici double pointer dec\n");
		double **res = new double*[twos];
		for (int i = 0; i < 8; i++) {
		res[i] = new double[3];
		}
		printf("global ici double pointer id\n");
		for (int i = 0; i < twos; i++) {
		for (int j = 0; j < 3; j++) {
		res[i][j] = cells[threadID][i][j];
		}
		}
		for (int i = 0; i < twos; i++) {
		for (int j = 0; j < 3; j++) {
		printf("global ici res : %f\n", res[i][j]);
		}
		}
	}*/
	/*if (threadID == 630) {
		for (int i = 630; i < 631; i++) {
			int twoS = cells[i][0][0];
			for (int j = 0; j < twoS; j++) {
				printf("\nres %d : %f		", i, cells[i][j][1]);
				printf("res %d : %f\n", i, cells[i][j][2]);
			}
			printf("\n");
		}
		printf("global ici 631.deger\n");
		for (int j = 0; j < points[631][0][3]; j++) {
			//std::cout << A[i][j][0] << " ";
			printf("X:%f		", points[631][j][0]);
			//std::cout << A[i][j][1] << " ";
			printf("Y:%f		", points[631][j][1]);
			//std::cout << A[i][j][3] << " ";
			printf("Z:%f		", points[631][j][2]);
			//std::cout << A[i][j][4] << " ";
			printf("S:%f		", points[631][j][3]);
			printf("\n");
		}
	}*/

	__global__ void spline(double ***points, double ***cells, double ***res, double ***mtx_l, double **mtx_v, int size)
	{
		int threadID = blockDim.x * blockIdx.x + threadIdx.x;
		//if (threadID > size) return;

		if (threadID < size)
		{
			int p = points[threadID][0][3];

			//double *mtx_v = new double[p + 3];

			/*if (threadID == 512) {
				printf("end dec\n");
			}*/
			//

			//double a = 0.0;

			for (int i = 0; i < p; ++i)
			{
				for (int j = i + 1; j < p; ++j)
				{
					//double pt_x = points[threadID][i][0] - points[threadID][j][0];
					//double pt_z = points[threadID][i][2] - points[threadID][j][2];
					
					//double x = pt_x * pt_x;
					//double z = pt_z * pt_z;
					double elen = sqrt(((points[threadID][i][0] - points[threadID][j][0])*(points[threadID][i][0] - points[threadID][j][0])) + ((points[threadID][i][2] - points[threadID][j][2])*(points[threadID][i][2] - points[threadID][j][2])));
					if (elen == 0)
					{
						mtx_l[threadID][i][j] = mtx_l[threadID][j][i] = 0.0;
					}
					else
					{
						mtx_l[threadID][i][j] = mtx_l[threadID][j][i] = elen * elen * log(elen);
					}
					//a += elen * 2;

					if (i == 0 && j == 1)
					{
						//printf("%f", elen);
					}
				}
			}

			//printf("first for\n");

			//a /= (double)(p * p);
			//printf("a : %f\n", a);
			for (int i = 0; i < p; ++i)
			{
				// diagonal: reqularization parameters (lambda * a^2)
				/*if (i == 0) {
					std::cout << "sirayla i mtx_l(0,1)" << i << mtx_l(0, 1) << std::endl;
				}*/
				mtx_l[threadID][i][i] = 0.0;

				// P (p x 3, upper right)
				mtx_l[threadID][i][p + 0] = 1.0;
				mtx_l[threadID][i][p + 1] = points[threadID][i][0];
				mtx_l[threadID][i][p + 2] = points[threadID][i][2];

				// P transposed (3 x p, bottom left)
				mtx_l[threadID][p + 0][i] = 1.0;
				mtx_l[threadID][p + 1][i] = points[threadID][i][0];
				mtx_l[threadID][p + 2][i] = points[threadID][i][2];
			}
			//printf("second for\n");

			// O (3 x 3, lower right)
			for (int i = p; i < p + 3; ++i)
				for (int j = p; j < p + 3; ++j)
					mtx_l[threadID][i][j] = 0.0;

			//printf("third for\n");
			// Fill the right hand vector V
			for (int i = 0; i < p; ++i)
				mtx_v[threadID][i] = points[threadID][i][1];
			mtx_v[threadID][p + 0] = mtx_v[threadID][p + 1] = mtx_v[threadID][p + 2] = 0.0;
			//printf("fourth for\n");

			/*if (threadID == 512) {
				printf("mtx_v\n");
				for (int i = 0; i < p+3; i++) {
					printf("%f\n", mtx_v[i]);
				}
				printf("mtx_l\n");
				for (int i = 0; i < p+3; i++) {
					for (int j = 0; j < p+3; j++) {
						printf("%f	", mtx_l[i][j]);
					}
					printf("\n");
				}
			}*/

			int m = p + 3, n = p + 3;

			//int pivsign = 0;
			int *piv = new int[m];

			for (int i = 0; i < m; ++i)
				piv[i] = i;
			//pivsign = 1;

			for (int j = 0; j < n; ++j)
			{
				double *col = new double[m];
				for (int i = 0; i < m; i++)
				{
					col[i] = mtx_l[threadID][i][j];
				}
				double *row = new double[n];
				for (int i = 0; i < m; ++i)
				{
					for (int l = 0; l < n; l++)
					{
						row[l] = mtx_l[threadID][i][l];
					}
					int kmax = fminf(i, j);
					double s = 0.0;
					for (int k = 0; k < kmax; k++)
					{
						s += row[k] * col[k];
					}
					row[j] = col[i] -= s;
					for (int l = 0; l < m; l++)
					{
						mtx_l[threadID][l][j] = col[l];
					}
					for (int l = 0; l < m; l++)
					{
						mtx_l[threadID][i][l] = row[l];
					}
				}
				free(row);

				int p = j;
				for (int i = j + 1; i < m; i++)
				{
					if (fabs(col[i]) > fabs(col[p]))
					{
						p = i;
					}
				}
				free(col);


				if (p != j)
				{
					for (int k = 0; k < n; k++)
					{
						double t = mtx_l[threadID][p][k];
						mtx_l[threadID][p][k] = mtx_l[threadID][j][k];
						mtx_l[threadID][j][k] = t;
					}
					int k = piv[p];
					piv[p] = piv[j];
					piv[j] = k;
					//pivsign = -pivsign;
				}

				if (j < m && mtx_l[threadID][j][j] != 0.0)
				{
					for (int i = j + 1; i < m; i++)
					{
						mtx_l[threadID][i][j] /= mtx_l[threadID][j][j];
					}
				}
			}

			double y = 0;
			for (int i = 0; i < m; ++i)
			{
				if (piv[i] != i)
				{
					y = mtx_v[threadID][i];
					mtx_v[threadID][i] = mtx_v[threadID][piv[i]];
					mtx_v[threadID][piv[i]] = y;
				}
				for (int j = i; j < m; ++j)
					if (piv[j] == i)
					{
						piv[j] = piv[i];
						break;
					}
			}
			free(piv);
			
			for (int k = 0; k < n; k++)
			{
				for (int i = k + 1; i < n; i++)
				{
					mtx_v[threadID][i] -= mtx_v[threadID][k] * mtx_l[threadID][i][k];
				}
			}

			/*printf("mtx_v\n");
			for (int i = 0; i < m; i++) {
				printf("%f\n", mtx_v[i]);
			}*/

			for (int k = n - 1; k >= 0; k--)
			{
				mtx_v[threadID][k] /= mtx_l[threadID][k][k];
				//printf("1. %f\n", mtx_v[k]);
				for (int i = 0; i < k; i++)
				{
					mtx_v[threadID][i] -= mtx_v[threadID][k] * mtx_l[threadID][i][k];
				}
				//printf("2. %f\n", mtx_v[k]);
			}
			/*printf("mtx_v\n");
			for (int i = 0; i < m; i++) {
				printf("%f\n", mtx_v[i]);
			}*/

			int trn = cells[threadID][0][0];
			//printf("trn : %d\n", trn);
			for (int j = 0; j < trn; j++)
			{
				/*printf("mtx_v[p+0] : %f", mtx_v[p + 0]);
				printf("mtx_v[p + 1] : %f", mtx_v[p + 1]);
				printf("cells[threadID][j][1] : %f", cells[threadID][j][1]);
				printf("mtx_v[p + 2] : %f", mtx_v[p + 2]);
				printf("cells[threadID][j][2] : %f", cells[threadID][j][2]);*/

				double h = mtx_v[threadID][p + 0] + mtx_v[threadID][p + 1] * cells[threadID][j][1] + mtx_v[threadID][p + 2] * cells[threadID][j][2];
				//printf("global ici h : %f\n", h);
				double cx = cells[threadID][j][1];
				double cy = cells[threadID][j][2];

				for (int i = 0; i < p; i++)
				{
					double x = points[threadID][i][0] - cx;
					double z = points[threadID][i][2] - cy;

					double xx = x * x;
					double zz = z * z;
					double elen = sqrtf(xx + zz);
					//printf("elen : %f  ", elen);
					//printf("sqrtf ile : %f  ", sqrtf((x*x) + (z*z)));

					if (elen == 0.0)
					{
						h += mtx_v[threadID][i] * 0.0;
					}
					else
					{
						h += mtx_v[threadID][i] * (elen * elen * logf(elen));
					}
					//printf("res h: %f\n", h);
				}
				//printf("\nson res h: %f", h);
				res[threadID][j][0] = h;
				//printf("\nres[threadID][j][0] = %f", res[threadID][j][0]);
				/*if (threadID == 0)
				{
					//printf("\n%d. thread h: %f res[0] : %f \n", threadID, h, res[threadID][j][0]);
					//res[threadID + 1][j][0] = res[threadID][j][0];
					//printf("\n%d. thread h: %f res[631] : %f \n", threadID + 1, h, res[threadID + 1][j][0]);
				}*/
			}
			//__syncthreads();
			free(mtx_v);
		}
	}

	void source3::cells(double ***A, double ***cellsize, double ***h_res, int Asize, int celS){
		//allocate control_points
		std::cout << "allocating mainArray	" << std::endl;
		double ***h_c = (double ***)malloc(Asize * sizeof(double **));
		for (int i = 0; i < Asize; i++)
		{
			int twoSize = A[i][0][3];
			h_c[i] = (double **)malloc(twoSize * sizeof(double *));
			for (int j = 0; j < twoSize; j++)
			{
				GPUerrchk(hipMalloc((void **)&h_c[i][j], 4 * sizeof(double)));
				GPUerrchk(hipMemcpy(h_c[i][j], A[i][j], 4 * sizeof(double), hipMemcpyHostToDevice));
			}
		}
		double ***h_c1 = (double ***)malloc(Asize * sizeof(double **));
		for (int i = 0; i < Asize; i++)
		{
			int twoSize = A[i][0][3];
			GPUerrchk(hipMalloc((void ***)&(h_c1[i]), twoSize * sizeof(double *)));
			GPUerrchk(hipMemcpy(h_c1[i], h_c[i], twoSize * sizeof(double *), hipMemcpyHostToDevice));
		}

		double ***d_c;
		GPUerrchk(hipMalloc((void ****)&d_c, Asize * sizeof(double **)));
		GPUerrchk(hipMemcpy(d_c, h_c1, Asize * sizeof(double **), hipMemcpyHostToDevice));

		//allocate cells
		std::cout << "allocating cellsArray" << std::endl;
		double ***cells1 = (double ***)malloc(Asize * sizeof(double **));
		for (int i = 0; i < Asize; i++)
		{
			int twoSize = cellsize[i][0][0];
			cells1[i] = (double **)malloc(twoSize * sizeof(double *));
			for (int j = 0; j < twoSize; j++)
			{
				GPUerrchk(hipMalloc((void **)&cells1[i][j], 3 * sizeof(double)));
				GPUerrchk(hipMemcpy(cells1[i][j], cellsize[i][j], 3 * sizeof(double), hipMemcpyHostToDevice));
			}
		}
		double ***cells = (double ***)malloc(Asize * sizeof(double **));
		for (int i = 0; i < Asize; i++)
		{
			int twoSize = cellsize[i][0][0];
			GPUerrchk(hipMalloc((void ***)&(cells[i]), twoSize * sizeof(double *)));
			GPUerrchk(hipMemcpy(cells[i], cells1[i], twoSize * sizeof(double *), hipMemcpyHostToDevice));
		}
		double ***d_cells;
		GPUerrchk(hipMalloc((void ****)&d_cells, Asize * sizeof(double **)));
		GPUerrchk(hipMemcpy(d_cells, cells, Asize * sizeof(double **), hipMemcpyHostToDevice));

		//allocate res
		std::cout << "allocating resArray" << std::endl;
		double ***res1 = (double ***)malloc(Asize * sizeof(double **));
		for (int i = 0; i < Asize; i++)
		{
			int twoSize = cellsize[i][0][0];
			res1[i] = (double **)malloc(twoSize * sizeof(double *));
			for (int j = 0; j < twoSize; j++)
			{
				GPUerrchk(hipMalloc((void **)&res1[i][j], 1 * sizeof(double)));
			}
		}
		double ***res2 = (double ***)malloc(Asize * sizeof(double **));
		for (int i = 0; i < Asize; i++)
		{
			int twoSize = cellsize[i][0][0];
			GPUerrchk(hipMalloc((void ***)&res2[i], twoSize * sizeof(double *)));
			GPUerrchk(hipMemcpy(res2[i], res1[i], twoSize * sizeof(double *), hipMemcpyHostToDevice));
		}
		double ***d_res;
		GPUerrchk(hipMalloc((void ****)&d_res, Asize * sizeof(double ***)));
		GPUerrchk(hipMemcpy(d_res, res2, Asize * sizeof(double **), hipMemcpyHostToDevice));

		std::cout << "allocating mtx_l" << std::endl;
		double ***mtx_l1 = (double ***)malloc(Asize * sizeof(double **));
		for (int i = 0; i < Asize; i++)
		{
			int twoSize = A[i][0][3] + 3;
			mtx_l1[i] = (double **)malloc(twoSize * sizeof(double *));
			for (int j = 0; j < twoSize; j++)
			{
				hipMalloc((void **)&mtx_l1[i][j], twoSize * sizeof(double));
			}
		}
		double ***mtx_l2 = (double ***)malloc(Asize * sizeof(double **));
		for (int i = 0; i < Asize; i++)
		{
			int twoSize = A[i][0][3] + 3;
			hipMalloc((void ***)&(mtx_l2[i]), twoSize * sizeof(double *));
			hipMemcpy(mtx_l2[i], mtx_l1[i], twoSize * sizeof(double *), hipMemcpyHostToDevice);
		}
		double ***d_mtx_l;
		hipMalloc((void ****)&d_mtx_l, Asize * sizeof(double **));
		hipMemcpy(d_mtx_l, mtx_l2, Asize * sizeof(double **), hipMemcpyHostToDevice);

		std::cout << "allocating mtx_v" << std::endl;
		double **mtx_v1 = (double **)malloc(Asize * sizeof(double **));
		for (int i = 0; i < Asize; i++)
		{
			int twoSize = A[i][0][3];
			hipMalloc(&mtx_v1[i], twoSize * sizeof(double));
		}
		double **d_mtx_v = (double **)malloc(Asize * sizeof(double **));
		hipMalloc(&d_mtx_v, Asize * sizeof(double **));
		hipMemcpy(d_mtx_v, mtx_v1, Asize * sizeof(double), hipMemcpyHostToDevice);


		unsigned int numberOfThreads = Asize;
		unsigned int requiredNumberOfBlocks = (numberOfThreads / 1024) + 1;
		dim3 block = dim3(1024, 1, 1);
		dim3 grid = dim3(requiredNumberOfBlocks, 1, 1);
		std::clock_t start;
		double duration;
		start = std::clock();
		//hipDeviceSetLimit(hipLimitMallocHeapSize, 32*1024*1024);
		printf("launch kernel	");
		spline<<<grid, block>>>(d_c, d_cells, d_res, d_mtx_l, d_mtx_v, Asize); //---->>>>>><<<<<<<>>>>>>><<<<<<<>>>>>>><<<<<<>>>>
		printf("end kernel \n");
		hipDeviceSynchronize();
		duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
		std::cout << "kernel process time :  " << duration << " seconds" << std::endl;
		std::cout << std::endl;


		printf("copy gpu to ram");
		for (int i = 0; i < Asize; i++)
		{
			int twoS = cellsize[i][0][0];
			for (int j = 0; j < twoS; j++)
			{
				hipMemcpy(&h_res[i][j][0], res1[i][j], 1 * sizeof(double), hipMemcpyDeviceToHost);
			}
		}

	}
}